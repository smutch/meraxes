#include "hip/hip_runtime.h"
//==============================================================================
//
// This code was developed as part of the Astronomy Data and Computing Services
// (ADACS; https://adacs.org.au) 2017B Software Support program.
//
// Written by: Gregory B. Poole
// Date:       September 2017
//
// It is distributed under the MIT (Expat) License (see https://opensource.org/):
//
// Copyright (c) 2017 Astronomy Data and Computing Services (ADACS)
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//
//==============================================================================

#include <complex.h>
#include <fftw3-mpi.h>
#include <fftw3.h>
#include <hdf5.h>
#include <hdf5_hl.h>
#include <math.h>

#include "XRayHeatingFunctions.h"
#include "find_HII_bubbles.h"
#include "meraxes.h"
#include "meraxes_gpu.hh"
#include "misc_tools.h"
#include "recombinations.h"
#include "reionization.h"

/*
 * ⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻
 * ------------------------------------------------------------
 * TODO LIST - @smutch (See also TODO items throughout the code)
 * ------------------------------------------------------------
 * - [X] Why are the unfiltered grids being copied to the GPU and then copied back each filtering step if using FFTW?
 * - [ ] Investigate the use of streams.
 * - [ ] The error handling is horrendously complicated. Is this really necessary?
 *
 * ⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻⎻
 */

/*
 * This code is a re-write of the modified version of 21cmFAST used in Mutch et
 * al. (2016; Meraxes paper).  The original code was written by Andrei Mesinger
 * with additions as detailed in Sobacchi & Mesinger (2013abc).  Updates were
 * subsequently made by Simon Mutch & Paul Geil.
 */

// This is the CUDA-enabled version of find_HII_bubbles().
void _find_HII_bubbles_gpu(const int snapshot, const bool flag_write_validation_output)
{
  // Fetch needed things from run_globals
  const MPI_Comm mpi_comm = run_globals.mpi_comm;
  const int mpi_rank = run_globals.mpi_rank;
  const double box_size = run_globals.params.BoxSize;
  const int ReionGridDim = run_globals.params.ReionGridDim;
  const int ReionUVBFlag = run_globals.params.ReionUVBFlag;
  const int Flag_IncludeRecombinations = run_globals.params.Flag_IncludeRecombinations;
  const double ReionEfficiency = run_globals.params.physics.ReionEfficiency;
  const double ReionNionPhotPerBary = run_globals.params.physics.ReionNionPhotPerBary;
  const double UnitLength_in_cm = run_globals.units.UnitLength_in_cm;
  const double UnitMass_in_g = run_globals.units.UnitMass_in_g;
  const double UnitTime_in_s = run_globals.units.UnitTime_in_s;
  const double ReionRBubbleMax = run_globals.params.Flag_IncludeRecombinations
                                   ? run_globals.params.physics.ReionRBubbleMaxRecomb
                                   : run_globals.params.physics.ReionRBubbleMax; // Mpc/h
  const double ReionRBubbleMin = run_globals.params.physics.ReionRBubbleMin;
  const double ReionDeltaRFactor = run_globals.params.ReionDeltaRFactor;
  const double ReionGammaHaloBias = run_globals.params.physics.ReionGammaHaloBias;
  const double ReionAlphaUV = run_globals.params.physics.ReionAlphaUV;
  const double Hubble_h = run_globals.params.Hubble_h;
  // const double ReionEscapeFrac = run_globals.params.physics.ReionEscapeFrac;
  // grid parameters
  const ptrdiff_t* slabs_nix = run_globals.reion_grids.slab_nix;
  const ptrdiff_t* slabs_n_complex = run_globals.reion_grids.slab_n_complex;
  const ptrdiff_t* slabs_ix_start = run_globals.reion_grids.slab_ix_start;
  const int local_nix = (int)slabs_nix[mpi_rank];
  const int local_ix_start = (int)slabs_ix_start[mpi_rank];
  const int slab_n_complex = (int)(slabs_n_complex[mpi_rank]);
  const int slab_n_real = local_nix * ReionGridDim * ReionGridDim;
  // preallocated grids
  float* J_21 = run_globals.reion_grids.J_21;         // real
  float* r_bubble = run_globals.reion_grids.r_bubble; // real
  // output grids
  float* xH = run_globals.reion_grids.xH;                                 // real
  float* z_at_ionization = run_globals.reion_grids.z_at_ionization;       // real
  float* J_21_at_ionization = run_globals.reion_grids.J_21_at_ionization; // real
  // output values
  double* volume_weighted_global_xH = &(run_globals.reion_grids.volume_weighted_global_xH);
  double* volume_weighted_global_J_21 = &(run_globals.reion_grids.volume_weighted_global_J_21);
  double* mass_weighted_global_xH = &(run_globals.reion_grids.mass_weighted_global_xH);
  // a few needed constants
  const double pixel_volume = pow(box_size / (double)ReionGridDim, 3); // (Mpc/h)^3
  const double total_n_cells = pow((double)ReionGridDim, 3);
  const double inv_total_n_cells = 1. / total_n_cells;

  const double redshift = run_globals.ZZ[snapshot];
  double prev_redshift;
  if (snapshot == 0) {
    prev_redshift = run_globals.ZZ[snapshot];
  } else {
    prev_redshift = run_globals.ZZ[snapshot - 1];
  }

  float zstep = (float)(prev_redshift - redshift);
  float fabs_dtdz = (float)fabs(dtdz((float)redshift) / Hubble_h);

  // Check that a valid filter option has been specified
  try {
    throw_on_generic_error(run_globals.params.ReionFilterType < 0 || run_globals.params.ReionFilterType > 2,
                           meraxes_cuda_exception::INVALID_FILTER);
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  // Initialize device arrays
  Complex* deltax_unfiltered_device = NULL;
  Complex* stars_unfiltered_device = NULL;
  Complex* weighted_sfr_unfiltered_device = NULL;
  Complex* deltax_filtered_device = NULL;
  Complex* stars_filtered_device = NULL;
  Complex* weighted_sfr_filtered_device = NULL;
  Complex* N_rec_unfiltered_device = NULL;
  Complex* N_rec_filtered_device = NULL;
  float* xH_device = NULL;
  float* r_bubble_device = NULL;
  float* z_at_ionization_device = NULL;
  float* J_21_at_ionization_device = NULL;
  float* J_21_device = NULL;
  float* Gamma12_device = NULL;

  try {
    throw_on_cuda_error(hipMalloc((void**)&deltax_unfiltered_device, sizeof(Complex) * slab_n_complex),
                        meraxes_cuda_exception::MALLOC);
    throw_on_cuda_error(hipMalloc((void**)&stars_unfiltered_device, sizeof(Complex) * slab_n_complex),
                        meraxes_cuda_exception::MALLOC);
    throw_on_cuda_error(hipMalloc((void**)&weighted_sfr_unfiltered_device, sizeof(Complex) * slab_n_complex),
                        meraxes_cuda_exception::MALLOC);
    throw_on_cuda_error(hipMalloc((void**)&deltax_filtered_device, sizeof(Complex) * slab_n_complex),
                        meraxes_cuda_exception::MALLOC);
    throw_on_cuda_error(hipMalloc((void**)&stars_filtered_device, sizeof(Complex) * slab_n_complex),
                        meraxes_cuda_exception::MALLOC);
    throw_on_cuda_error(hipMalloc((void**)&weighted_sfr_filtered_device, sizeof(Complex) * slab_n_complex),
                        meraxes_cuda_exception::MALLOC);

    if (Flag_IncludeRecombinations) {
      throw_on_cuda_error(hipMalloc((void**)&N_rec_unfiltered_device, sizeof(Complex) * slab_n_complex),
                          meraxes_cuda_exception::MALLOC);
      throw_on_cuda_error(hipMalloc((void**)&N_rec_filtered_device, sizeof(Complex) * slab_n_complex),
                          meraxes_cuda_exception::MALLOC);
    }

    if (slab_n_real > 0) {
      throw_on_cuda_error(hipMalloc((void**)&xH_device, sizeof(float) * slab_n_real), meraxes_cuda_exception::MALLOC);
      throw_on_cuda_error(hipMalloc((void**)&r_bubble_device, sizeof(float) * slab_n_real),
                          meraxes_cuda_exception::MALLOC);
      throw_on_cuda_error(hipMalloc((void**)&z_at_ionization_device, sizeof(float) * slab_n_real),
                          meraxes_cuda_exception::MALLOC);
      throw_on_cuda_error(hipMalloc((void**)&J_21_at_ionization_device, sizeof(float) * slab_n_real),
                          meraxes_cuda_exception::MALLOC);
      if (ReionUVBFlag)
        throw_on_cuda_error(hipMalloc((void**)&J_21_device, sizeof(float) * slab_n_real),
                            meraxes_cuda_exception::MALLOC);
      if (Flag_IncludeRecombinations) {
        throw_on_cuda_error(hipMalloc((void**)&Gamma12_device, sizeof(float) * slab_n_real),
                            meraxes_cuda_exception::MALLOC);
      }
    }

    // Throw an exception if another rank has thrown one
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  float* deltax = run_globals.reion_grids.deltax;
  fftwf_complex* deltax_unfiltered = run_globals.reion_grids.deltax_unfiltered;
  Complex* deltax_filtered = (Complex*)run_globals.reion_grids.deltax_filtered;
  fftwf_execute(run_globals.reion_grids.deltax_forward_plan);

  fftwf_complex* stars_unfiltered = run_globals.reion_grids.stars_unfiltered;
  Complex* stars_filtered = (Complex*)run_globals.reion_grids.stars_filtered;
  fftwf_execute(run_globals.reion_grids.stars_forward_plan);

  fftwf_complex* weighted_sfr_unfiltered = run_globals.reion_grids.weighted_sfr_unfiltered;
  Complex* weighted_sfr_filtered = (Complex*)run_globals.reion_grids.weighted_sfr_filtered;
  fftwf_execute(run_globals.reion_grids.weighted_sfr_forward_plan);

  float* Gamma12 = run_globals.reion_grids.Gamma12;
  float* N_rec = run_globals.reion_grids.N_rec;
  fftwf_complex* N_rec_unfiltered = NULL;
  Complex* N_rec_filtered = NULL;
  if (Flag_IncludeRecombinations) {
    N_rec_unfiltered = run_globals.reion_grids.N_rec_unfiltered;
    N_rec_filtered = (Complex*)run_globals.reion_grids.N_rec_filtered;
    fftwf_execute(run_globals.reion_grids.N_rec_forward_plan);
  }

  // Perform host -> device transfer of input grids.
  try {
    throw_on_cuda_error(
      hipMemcpy(
        deltax_unfiltered_device, deltax_unfiltered, sizeof(float) * 2 * slab_n_complex, hipMemcpyHostToDevice),
      meraxes_cuda_exception::MEMCPY);
    throw_on_cuda_error(
      hipMemcpy(stars_unfiltered_device, stars_unfiltered, sizeof(float) * 2 * slab_n_complex, hipMemcpyHostToDevice),
      meraxes_cuda_exception::MEMCPY);
    throw_on_cuda_error(hipMemcpy(weighted_sfr_unfiltered_device,
                                   weighted_sfr_unfiltered,
                                   sizeof(float) * 2 * slab_n_complex,
                                   hipMemcpyHostToDevice),
                        meraxes_cuda_exception::MEMCPY);

    if (Flag_IncludeRecombinations)
      throw_on_cuda_error(
        hipMemcpy(
          N_rec_unfiltered_device, N_rec_unfiltered, sizeof(float) * 2 * slab_n_complex, hipMemcpyHostToDevice),
        meraxes_cuda_exception::MEMCPY);

    if (slab_n_real) {
      throw_on_cuda_error(
        hipMemcpy(z_at_ionization_device, z_at_ionization, sizeof(float) * slab_n_real, hipMemcpyHostToDevice),
        meraxes_cuda_exception::MEMCPY);
      throw_on_cuda_error(
        hipMemcpy(J_21_at_ionization_device, J_21_at_ionization, sizeof(float) * slab_n_real, hipMemcpyHostToDevice),
        meraxes_cuda_exception::MEMCPY);

      if (Flag_IncludeRecombinations) {
        throw_on_cuda_error(hipMemcpy(Gamma12_device, Gamma12, sizeof(float) * slab_n_real, hipMemcpyHostToDevice),
                            meraxes_cuda_exception::MEMCPY);
      }
    }

    // Throw an exception if another rank has thrown one
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  // Initialize GPU block and thread count
  int threads = run_globals.gpu->n_threads;
  int grid_complex = (slab_n_complex + (threads - 1)) / threads;
  int grid_real = (slab_n_real + (threads - 1)) / threads;

  // mlog("threads = %d", MLOG_ALLRANKS|MLOG_MESG, threads);
  // mlog("slab_n_complex = %d", MLOG_ALLRANKS|MLOG_MESG, slab_n_complex);
  // mlog("grid_complex = %d", MLOG_ALLRANKS|MLOG_MESG, grid_complex);
  // mlog("slab_n_real = %d", MLOG_ALLRANKS|MLOG_MESG, slab_n_real);
  // mlog("grid_real = %d", MLOG_ALLRANKS|MLOG_MESG|MLOG_FLUSH, grid_real);

  // MPI_Barrier(run_globals.mpi_comm);

  // Remember to add the factor of VOLUME/TOT_NUM_PIXELS when converting from real space to k-space
  // Note: we will leave off factor of VOLUME, in anticipation of the inverse FFT below
  try {
    throw_on_kernel_error((complex_vector_times_scalar<<<grid_complex, threads>>>(
                            deltax_unfiltered_device, inv_total_n_cells, slab_n_complex)),
                          meraxes_cuda_exception::KERNEL_CMPLX_AX);
    throw_on_kernel_error((complex_vector_times_scalar<<<grid_complex, threads>>>(
                            stars_unfiltered_device, inv_total_n_cells, slab_n_complex)),
                          meraxes_cuda_exception::KERNEL_CMPLX_AX);
    throw_on_kernel_error((complex_vector_times_scalar<<<grid_complex, threads>>>(
                            weighted_sfr_unfiltered_device, inv_total_n_cells, slab_n_complex)),
                          meraxes_cuda_exception::KERNEL_CMPLX_AX);

    if (Flag_IncludeRecombinations)
      throw_on_kernel_error((complex_vector_times_scalar<<<grid_complex, threads>>>(
                              N_rec_unfiltered_device, inv_total_n_cells, slab_n_complex)),
                            meraxes_cuda_exception::KERNEL_CMPLX_AX);

    check_thread_sync(meraxes_cuda_exception::KERNEL_CMPLX_AX);
    // Throw an exception if another rank has thrown one
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  // Initialize a few of the output grids
  try {
    if (slab_n_real > 0) {
      throw_on_kernel_error((set_array_gpu<<<grid_real, threads>>>(xH_device, slab_n_real, 1.f)),
                            meraxes_cuda_exception::KERNEL_SET_ARRAY);
      throw_on_kernel_error((set_array_gpu<<<grid_real, threads>>>(r_bubble_device, slab_n_real, 0.f)),
                            meraxes_cuda_exception::KERNEL_SET_ARRAY);
      if (ReionUVBFlag)
        throw_on_kernel_error((set_array_gpu<<<grid_real, threads>>>(J_21_device, slab_n_real, 0.f)),
                              meraxes_cuda_exception::KERNEL_SET_ARRAY);
    }
    check_thread_sync(meraxes_cuda_exception::KERNEL_SET_ARRAY);
    // Throw an exception if another rank has thrown one
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  // This parameter choice is sensitive to noise on the cell size, at least for the typical
  // cell sizes in RT simulations. It probably doesn't matter for larger cell sizes.
  double cell_length_factor = L_FACTOR;
  if ((box_size / (double)ReionGridDim) < 1.0) // Fairly arbitrary length based on 2 runs Sobacchi did
    cell_length_factor = 1.0;

  // Loop through filter radii
  double R = fmin(ReionRBubbleMax, L_FACTOR * box_size); // Mpc/h
  bool flag_last_filter_step = false;
  int i_R = 0;
  while (!flag_last_filter_step) {
    i_R++;

    // check to see if this is our last filtering step
    if (((R / ReionDeltaRFactor) <= (cell_length_factor * box_size / (double)ReionGridDim)) ||
        ((R / ReionDeltaRFactor) <= ReionRBubbleMin)) {
      flag_last_filter_step = true;
      R = cell_length_factor * box_size / (double)ReionGridDim;
    }

    mlog(".", MLOG_CONT);

    // Create working copies of the k-space grids
    try {
      throw_on_cuda_error(
        hipMemcpy(
          deltax_filtered_device, deltax_unfiltered_device, sizeof(Complex) * slab_n_complex, hipMemcpyDeviceToDevice),
        meraxes_cuda_exception::MEMCPY);
      throw_on_cuda_error(
        hipMemcpy(
          stars_filtered_device, stars_unfiltered_device, sizeof(Complex) * slab_n_complex, hipMemcpyDeviceToDevice),
        meraxes_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy(weighted_sfr_filtered_device,
                                     weighted_sfr_unfiltered_device,
                                     sizeof(Complex) * slab_n_complex,
                                     hipMemcpyDeviceToDevice),
                          meraxes_cuda_exception::MEMCPY);

      if (Flag_IncludeRecombinations)
        throw_on_cuda_error(
          hipMemcpy(
            N_rec_filtered_device, N_rec_unfiltered_device, sizeof(Complex) * slab_n_complex, hipMemcpyDeviceToDevice),
          meraxes_cuda_exception::MEMCPY);

      // Throw an exception if another rank has thrown one
      throw_on_global_error();
    } catch (const meraxes_cuda_exception e) {
      e.process_exception();
    }

    // Perform convolution
    if (!flag_last_filter_step) {
      try {
        throw_on_kernel_error((filter_gpu<<<grid_complex, threads>>>(deltax_filtered_device,
                                                                     ReionGridDim,
                                                                     local_ix_start,
                                                                     slab_n_complex,
                                                                     R,
                                                                     box_size,
                                                                     run_globals.params.ReionFilterType)),
                              meraxes_cuda_exception::KERNEL_FILTER);
        throw_on_kernel_error((filter_gpu<<<grid_complex, threads>>>(stars_filtered_device,
                                                                     ReionGridDim,
                                                                     local_ix_start,
                                                                     slab_n_complex,
                                                                     R,
                                                                     box_size,
                                                                     run_globals.params.ReionFilterType)),
                              meraxes_cuda_exception::KERNEL_FILTER);
        throw_on_kernel_error((filter_gpu<<<grid_complex, threads>>>(weighted_sfr_filtered_device,
                                                                     ReionGridDim,
                                                                     local_ix_start,
                                                                     slab_n_complex,
                                                                     R,
                                                                     box_size,
                                                                     run_globals.params.ReionFilterType)),
                              meraxes_cuda_exception::KERNEL_FILTER);

        if (Flag_IncludeRecombinations)
          throw_on_kernel_error((filter_gpu<<<grid_complex, threads>>>(N_rec_filtered_device,
                                                                       ReionGridDim,
                                                                       local_ix_start,
                                                                       slab_n_complex,
                                                                       R,
                                                                       box_size,
                                                                       run_globals.params.ReionFilterType)),
                                meraxes_cuda_exception::KERNEL_FILTER);

        check_thread_sync(meraxes_cuda_exception::KERNEL_FILTER);
        // Throw an exception if another rank has thrown one
        throw_on_global_error();
      } catch (const meraxes_cuda_exception e) {
        e.process_exception();
      }
    }

    // inverse fourier transform back to real space
    try {
      throw_on_cuda_error(
        hipMemcpy(deltax_filtered, deltax_filtered_device, sizeof(float) * 2 * slab_n_complex, hipMemcpyDeviceToHost),
        meraxes_cuda_exception::MEMCPY);
      fftwf_execute(run_globals.reion_grids.deltax_filtered_reverse_plan);
      throw_on_cuda_error(
        hipMemcpy(deltax_filtered_device, deltax_filtered, sizeof(float) * 2 * slab_n_complex, hipMemcpyHostToDevice),
        meraxes_cuda_exception::MEMCPY);

      throw_on_cuda_error(
        hipMemcpy(stars_filtered, stars_filtered_device, sizeof(float) * 2 * slab_n_complex, hipMemcpyDeviceToHost),
        meraxes_cuda_exception::MEMCPY);
      fftwf_execute(run_globals.reion_grids.stars_filtered_reverse_plan);
      throw_on_cuda_error(
        hipMemcpy(stars_filtered_device, stars_filtered, sizeof(float) * 2 * slab_n_complex, hipMemcpyHostToDevice),
        meraxes_cuda_exception::MEMCPY);

      throw_on_cuda_error(hipMemcpy(weighted_sfr_filtered,
                                     weighted_sfr_filtered_device,
                                     sizeof(float) * 2 * slab_n_complex,
                                     hipMemcpyDeviceToHost),
                          meraxes_cuda_exception::MEMCPY);
      fftwf_execute(run_globals.reion_grids.weighted_sfr_filtered_reverse_plan);
      throw_on_cuda_error(hipMemcpy(weighted_sfr_filtered_device,
                                     weighted_sfr_filtered,
                                     sizeof(float) * 2 * slab_n_complex,
                                     hipMemcpyHostToDevice),
                          meraxes_cuda_exception::MEMCPY);

      if (Flag_IncludeRecombinations) {
        throw_on_cuda_error(
          hipMemcpy(N_rec_filtered, N_rec_filtered_device, sizeof(float) * 2 * slab_n_complex, hipMemcpyDeviceToHost),
          meraxes_cuda_exception::MEMCPY);
        fftwf_execute(run_globals.reion_grids.N_rec_filtered_reverse_plan);
        throw_on_cuda_error(
          hipMemcpy(N_rec_filtered_device, N_rec_filtered, sizeof(float) * 2 * slab_n_complex, hipMemcpyHostToDevice),
          meraxes_cuda_exception::MEMCPY);
      }
      // Throw an exception if another rank has thrown one
      throw_on_global_error();
    } catch (const meraxes_cuda_exception e) {
      e.process_exception();
    }

    // Perform sanity checks to account for aliasing effects
    try {
      if (slab_n_real > 0) {
        throw_on_kernel_error((sanity_check_aliasing<<<grid_real, threads>>>(
                                deltax_filtered_device, ReionGridDim, slab_n_real, -1.f + REL_TOL)),
                              meraxes_cuda_exception::KERNEL_CHECK);
        throw_on_kernel_error(
          (sanity_check_aliasing<<<grid_real, threads>>>(stars_filtered_device, ReionGridDim, slab_n_real, 0.f)),
          meraxes_cuda_exception::KERNEL_CHECK);
        throw_on_kernel_error(
          (sanity_check_aliasing<<<grid_real, threads>>>(weighted_sfr_filtered_device, ReionGridDim, slab_n_real, 0.f)),
          meraxes_cuda_exception::KERNEL_CHECK);

        if (Flag_IncludeRecombinations)
          throw_on_kernel_error(
            (sanity_check_aliasing<<<grid_real, threads>>>(N_rec_filtered_device, ReionGridDim, slab_n_real, 0.f)),
            meraxes_cuda_exception::KERNEL_CHECK);
      }
      check_thread_sync(meraxes_cuda_exception::KERNEL_CHECK);

      // Throw an exception if another rank has thrown one
      throw_on_global_error();
    } catch (const meraxes_cuda_exception e) {
      e.process_exception();
    }

    // Main loop through the box...
    const double J_21_aux_constant = (1.0 + redshift) * (1.0 + redshift) / (4.0 * M_PI) * ReionAlphaUV * PLANCK *
                                     1e21 // * ReionEscapeFrac
                                     * R * UnitLength_in_cm * ReionNionPhotPerBary / PROTONMASS * UnitMass_in_g /
                                     pow(UnitLength_in_cm, 3) / UnitTime_in_s;
    const double inv_pixel_volume = 1. / pixel_volume;

    double Gamma_R_prefactor = 1.0;
    if (Flag_IncludeRecombinations) {
      Gamma_R_prefactor = (1.0 + redshift) * (1.0 + redshift) * R * (UnitLength_in_cm / Hubble_h) * SIGMA_HI *
                          ReionAlphaUV / (ReionAlphaUV + 2.75) / 1.0e-12; // Converting R h^-1 to R.
    }

    try {
      if (slab_n_real > 0) {
        throw_on_kernel_error((find_HII_bubbles_gpu_main_loop<<<grid_real, threads>>>(redshift,
                                                                                      slab_n_real,
                                                                                      flag_last_filter_step,
                                                                                      ReionUVBFlag,
                                                                                      Flag_IncludeRecombinations,
                                                                                      ReionGridDim,
                                                                                      R,
                                                                                      RtoM(R),
                                                                                      ReionEfficiency,
                                                                                      inv_pixel_volume,
                                                                                      J_21_aux_constant,
                                                                                      ReionGammaHaloBias,
                                                                                      UnitMass_in_g,
                                                                                      UnitTime_in_s,
                                                                                      UnitLength_in_cm,
                                                                                      Hubble_h,
                                                                                      ReionNionPhotPerBary,
                                                                                      Gamma_R_prefactor,
                                                                                      xH_device,
                                                                                      J_21_device,
                                                                                      r_bubble_device,
                                                                                      J_21_at_ionization_device,
                                                                                      z_at_ionization_device,
                                                                                      Gamma12_device,
                                                                                      deltax_filtered_device,
                                                                                      stars_filtered_device,
                                                                                      weighted_sfr_filtered_device,
                                                                                      N_rec_unfiltered_device)),
                              meraxes_cuda_exception::KERNEL_MAIN_LOOP);
      }
      check_thread_sync(meraxes_cuda_exception::KERNEL_MAIN_LOOP);
      // Throw an exception if another rank has thrown one
      throw_on_global_error();
    } catch (const meraxes_cuda_exception e) {
      e.process_exception();
    }

    R /= ReionDeltaRFactor;
  }

  // Perform device -> host transfer
  try {
    if (slab_n_real > 0) {
      throw_on_cuda_error(hipMemcpy((void*)xH, (void*)xH_device, sizeof(float) * slab_n_real, hipMemcpyDeviceToHost),
                          meraxes_cuda_exception::MEMCPY);
      throw_on_cuda_error(
        hipMemcpy((void*)r_bubble, (void*)r_bubble_device, sizeof(float) * slab_n_real, hipMemcpyDeviceToHost),
        meraxes_cuda_exception::MEMCPY);
      if (ReionUVBFlag)
        throw_on_cuda_error(
          hipMemcpy((void*)J_21, (void*)J_21_device, sizeof(float) * slab_n_real, hipMemcpyDeviceToHost),
          meraxes_cuda_exception::MEMCPY);
      throw_on_cuda_error(
        hipMemcpy(
          (void*)z_at_ionization, (void*)z_at_ionization_device, sizeof(float) * slab_n_real, hipMemcpyDeviceToHost),
        meraxes_cuda_exception::MEMCPY);
      throw_on_cuda_error(hipMemcpy((void*)J_21_at_ionization,
                                     (void*)J_21_at_ionization_device,
                                     sizeof(float) * slab_n_real,
                                     hipMemcpyDeviceToHost),
                          meraxes_cuda_exception::MEMCPY);

      if (Flag_IncludeRecombinations) {
        throw_on_cuda_error(
          hipMemcpy((void*)Gamma12, (void*)Gamma12_device, sizeof(float) * slab_n_real, hipMemcpyDeviceToHost),
          meraxes_cuda_exception::MEMCPY);
      }
    }
    throw_on_cuda_error(
      hipMemcpy(
        (void*)deltax, (void*)deltax_filtered_device, sizeof(float) * 2 * slab_n_complex, hipMemcpyDeviceToHost),
      meraxes_cuda_exception::MEMCPY);
    // Throw an exception if another rank has thrown one
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  // Clean-up device
  try {
    throw_on_cuda_error(hipFree(deltax_unfiltered_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(stars_unfiltered_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(weighted_sfr_unfiltered_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(deltax_filtered_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(stars_filtered_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(weighted_sfr_filtered_device), meraxes_cuda_exception::FREE);

    if (Flag_IncludeRecombinations) {
      throw_on_cuda_error(hipFree(N_rec_unfiltered_device), meraxes_cuda_exception::FREE);
      throw_on_cuda_error(hipFree(N_rec_filtered_device), meraxes_cuda_exception::FREE);
    }

    throw_on_cuda_error(hipFree(xH_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(r_bubble_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(z_at_ionization_device), meraxes_cuda_exception::FREE);
    throw_on_cuda_error(hipFree(J_21_at_ionization_device), meraxes_cuda_exception::FREE);

    if (ReionUVBFlag)
      throw_on_cuda_error(hipFree(J_21_device), meraxes_cuda_exception::FREE);

    if (Flag_IncludeRecombinations) {
      throw_on_cuda_error(hipFree(Gamma12_device), meraxes_cuda_exception::FREE);
    }

    // Throw an exception if another rank has thrown one
    throw_on_global_error();
  } catch (const meraxes_cuda_exception e) {
    e.process_exception();
  }

  // Find the volume and mass weighted neutral fractions
  // TODO: The deltax grid will have rounding errors from forward and reverse
  //       FFT. Should cache deltax slabs prior to ffts and reuse here.
  *volume_weighted_global_xH = 0.0;
  *volume_weighted_global_J_21 = 0.0;
  *mass_weighted_global_xH = 0.0;
  double mass_weight = 0.0;

  // Calculate neutral fractions.
  // TODO: A parallel reduction could be done for this before results are off-loaded from the GPU.
  //       This would require figuring out how to do the spline interpolation on the GPU though...
  int ix, iy, iz;
  for (ix = 0; ix < local_nix; ix++)
    for (iy = 0; iy < ReionGridDim; iy++)
      for (iz = 0; iz < ReionGridDim; iz++) {
        const int i_real = grid_index(ix, iy, iz, ReionGridDim, INDEX_REAL);
        const int i_padded = grid_index(ix, iy, iz, ReionGridDim, INDEX_PADDED);
        const double density_over_mean = 1.0 + (double)((float*)deltax)[i_padded];
        const double cell_xH = (double)(xH[i_real]);
        *volume_weighted_global_xH += cell_xH;
        *volume_weighted_global_J_21 += (double)J_21[i_real];
        *mass_weighted_global_xH += cell_xH * density_over_mean;
        mass_weight += density_over_mean;

        if (Flag_IncludeRecombinations) {
          const float z_eff = (float)((1. + redshift) * pow(density_over_mean, 1.0 / 3.0) - 1);
          const float dNrec = splined_recombination_rate(z_eff, Gamma12[i_real]) * fabs_dtdz * zstep * (1. - cell_xH);
          N_rec[i_padded] += dNrec;
        }
      }
  MPI_Allreduce(MPI_IN_PLACE, volume_weighted_global_xH, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);
  MPI_Allreduce(MPI_IN_PLACE, volume_weighted_global_J_21, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);
  MPI_Allreduce(MPI_IN_PLACE, mass_weighted_global_xH, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);
  MPI_Allreduce(MPI_IN_PLACE, &mass_weight, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);
  *volume_weighted_global_xH *= inv_total_n_cells;
  *volume_weighted_global_J_21 *= inv_total_n_cells;
  *mass_weighted_global_xH /= mass_weight;
}

// vim:set et sw=2 ts=2:
